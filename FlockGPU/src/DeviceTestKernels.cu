#include "hip/hip_runtime.h"
#include "DeviceTestKernels.cuh"
#include "Hash.cuh"
#include "FlockSystem.h"

__global__ void callGridFromPoint( int3 *_gridIdx, float3 *_pt)
{
    uint gIdx = blockIdx.x *blockDim.x + threadIdx.x;
    _gridIdx[gIdx] = gridFromPoint(_pt[gIdx]);
}
__global__ void callCellFromGrid( uint *_cellIdx, int3 *_grid)
{
    uint gIdx = blockIdx.x *blockDim.x + threadIdx.x;

    _cellIdx[gIdx] = cellFromGrid(_grid[gIdx]);
}
__global__ void callDist2( float *_dist2, float3 *_pos1,  float3 *_pos2)
{
    uint gIdx = blockIdx.x *blockDim.x + threadIdx.x;
    _dist2[gIdx] = dist2(_pos1[gIdx],_pos2[gIdx]);
}
__global__ void callRotateZ( float3 *_rot, float3 *_v,  float *_angle)
{
    uint gIdx = blockIdx.x *blockDim.x + threadIdx.x;
    _rot[gIdx] = rotateZ(_v[gIdx],_angle[gIdx] * 360.0f * RADIANS_F);

}

__global__ void callResolveForce(float3 *_pos, float3 *_v, const float3 *_f, const float _vMax)
{
    uint gIdx = blockIdx.x *blockDim.x + threadIdx.x;
    //resolveForce(_pos[gIdx], _v[gIdx], _f[gIdx], _vMax);
    resolveForce(_pos[gIdx], _v[gIdx], _f[gIdx], _vMax);
}


__global__ void callWander( float3 *_target, float *_angle,  float3 *_v,  float3 *_pos)
{
    uint gIdx = blockIdx.x *blockDim.x + threadIdx.x;
    _target[gIdx] = boidWanderPattern(_angle[gIdx], _v[gIdx], _pos[gIdx]);
}

__global__ void callSeek(  float3 *_f, float3  *_pos,  float3  *_v,  float3 *_target, const float _vMax)
{
    uint gIdx = blockIdx.x *blockDim.x + threadIdx.x;
    _f[gIdx] = boidSeekPattern(_pos[gIdx], _v[gIdx], _target[gIdx], _vMax);
}

__global__ void callFlee(  float3 *_f, float3  *_pos,  float3  *_v,  float3  *_target, const float _vMax)
{
    uint gIdx = blockIdx.x *blockDim.x + threadIdx.x;
    _f[gIdx] = boidFleePattern(_pos[gIdx], _v[gIdx], _target[gIdx], _vMax);
}




void testGridFromPoint(thrust::device_vector<int3> &_gridIdx, thrust::device_vector<float3> &_pt)
{
    int3 * grid = thrust::raw_pointer_cast(&_gridIdx[0]);
    float3 * pos = thrust::raw_pointer_cast(&_pt[0]);

    callGridFromPoint<<<1, _gridIdx.size()>>>(grid, pos);
    hipDeviceSynchronize();
}
void testCellFromGrid( thrust::device_vector<uint> &_cellIdx, thrust::device_vector<int3> &_grid)
{
    uint * cell = thrust::raw_pointer_cast(&_cellIdx[0]);
    int3 * grid = thrust::raw_pointer_cast(&_grid[0]);
    callCellFromGrid<<<1, _cellIdx.size()>>>(cell, grid);
    hipDeviceSynchronize();
}
void testDist2( thrust::device_vector<float> &_dist2, thrust::device_vector<float3> &_pos1,  thrust::device_vector<float3> &_pos2)
{
    float * dist2 = thrust::raw_pointer_cast(&_dist2[0]);
    float3 * pos1 = thrust::raw_pointer_cast(&_pos1[0]);
    float3 * pos2 = thrust::raw_pointer_cast(&_pos2[0]);
    callDist2<<<1, _dist2.size()>>>(dist2, pos1, pos2);
    hipDeviceSynchronize();
}
void testRotateZ( thrust::device_vector<float3> &_rot, thrust::device_vector<float3> &_v,  thrust::device_vector<float> &_angle)
{
    float3 * rot = thrust::raw_pointer_cast(&_rot[0]);
    float3 * v = thrust::raw_pointer_cast(&_v[0]);
    float * angle = thrust::raw_pointer_cast(&_angle[0]);
    callRotateZ<<<1, _rot.size()>>>(rot, v, angle);
    hipDeviceSynchronize();
}
void testResolveForce(
    thrust::device_vector<float3> &_pos, 
    thrust::device_vector<float3> &_v, 
    thrust::device_vector<float3> &_f, 
    float &_vMax)
{
    float3 * pos = thrust::raw_pointer_cast(&_pos[0]);
    float3 * v = thrust::raw_pointer_cast(&_v[0]);
    float3 * f = thrust::raw_pointer_cast(&_f[0]);

    thrust::device_ptr<float> vMax(&_vMax);

    callResolveForce<<<1, _pos.size()>>>( pos, v, f, *vMax.get());
    hipDeviceSynchronize();

}

void testWander( 
    thrust::device_vector<float3> &_target, 
    thrust::device_vector<float> &_angle, 
    thrust::device_vector<float3> &_v, 
    thrust::device_vector<float3 >&_pos)
{
    float3 * target= thrust::raw_pointer_cast(&_target[0]);
    float * angle = thrust::raw_pointer_cast(&_angle[0]);
    float3 * v = thrust::raw_pointer_cast(&_v[0]);
    float3 * pos = thrust::raw_pointer_cast(&_pos[0]);
    callWander<<<1, _target.size()>>>( target, angle, v, pos);
    hipDeviceSynchronize();
}
void testSeek( 
    thrust::device_vector<float3> &_f, 
    thrust::device_vector<float3> &_pos, 
    thrust::device_vector<float3> &_v, 
    thrust::device_vector<float3> &_target,  
    float &_vMax)
{
    float3 * f = thrust::raw_pointer_cast(&_f[0]);
    float3 * pos = thrust::raw_pointer_cast(&_pos[0]);
    float3 * v = thrust::raw_pointer_cast(&_v[0]);
    float3 * target = thrust::raw_pointer_cast(&_target[0]);

    thrust::device_ptr<float> vMax(&_vMax);
    //float * vMaxPtr = thrust::raw_pointer_cast(vMax.get());
    callSeek<<<1, _f.size()>>>(f, pos, v, target, *vMax.get());
    hipDeviceSynchronize();
}
void testFlee( 
    thrust::device_vector<float3> &_f, 
    thrust::device_vector<float3> &_pos, 
    thrust::device_vector<float3> &_v, 
    thrust::device_vector<float3> &_target, 
    float &_vMax)
{
    float3 * f = thrust::raw_pointer_cast(&_f[0]);
    float3 * pos = thrust::raw_pointer_cast(&_pos[0]);
    float3 * v = thrust::raw_pointer_cast(&_v[0]);
    float3 * target = thrust::raw_pointer_cast(&_target[0]);

    thrust::device_ptr<float> vMax(&_vMax);
    //float * vMaxPtr = thrust::raw_pointer_cast(vMax.get());
    callFlee<<<1, _f.size()>>>(f, pos, v, target, *vMax.get());
    hipDeviceSynchronize();
}

void testNeighbour(
    const float &_dt,
    const uint &_numP,
    const float &_res)
{
    FlockSystem flockSys(_numP,10.0f,0.1f,_dt,_res);
    flockSys.init();
    thrust::device_vector<uint> d_cellOcc(_numP);d_cellOcc = flockSys.getCellOcc();
    thrust::device_vector<uint> d_scatter(_numP);d_scatter = flockSys.getScatterAddress();
    thrust::device_vector<uint> d_hash(_numP);d_hash = flockSys.getHash();
    thrust::device_vector<bool> d_collision(_numP);d_collision = flockSys.getCollisionFlag();
    thrust::device_vector<float3> d_pos(_numP); d_pos= flockSys.getPos();
    thrust::device_vector<float3> d_target(_numP);d_target = flockSys.getTarget();

    float3 * pos = thrust::raw_pointer_cast(&d_pos[0]);
    float3 * targetPos = thrust::raw_pointer_cast(&d_target[0]);
    bool * collision = thrust::raw_pointer_cast(&d_collision[0]);
    uint * cellOcc = thrust::raw_pointer_cast(&d_cellOcc[0]);
    uint * scatter = thrust::raw_pointer_cast(&d_scatter[0]);

    thrust::fill(d_cellOcc.begin(), d_cellOcc.end(), 0);
    PointHashOperator hashOp(cellOcc);
    thrust::transform(d_pos.begin(), d_pos.end(), d_hash.begin(), hashOp);

    thrust::sort_by_key(
        d_hash.begin(),
        d_hash.end(),
        thrust::make_zip_iterator(thrust::make_tuple(d_pos.begin(),
                                                     d_target.begin()
        )));

    thrust::exclusive_scan(d_cellOcc.begin(), d_cellOcc.end(), d_scatter.begin());
    uint maxCellOcc = thrust::reduce(d_cellOcc.begin(), d_cellOcc.end(), 0, thrust::maximum<unsigned int>());
    uint blockSize = 32 * ceil(maxCellOcc / 32.0f);
    dim3 gridSize = dim3(_res, _res);


    computeAvgNeighbourPos<<<gridSize, blockSize>>>(collision, targetPos, pos, cellOcc, scatter);
    hipDeviceSynchronize();

}

void testHash(
    const float &_dt,
    const uint &_numP,
    const float &_res)                                                                
    {                                                                                                           
        FlockSystem flockSys(_numP,10.0f,0.1f,_dt,_res);                                                           
        flockSys.init();                                                                                        
        thrust::device_vector<uint> d_cellOcc = flockSys.getCellOcc();                                          
        thrust::device_vector<uint> d_scatter = flockSys.getScatterAddress();                                   
        thrust::device_vector<uint> d_hash = flockSys.getHash();                                                
        thrust::device_vector<float3> d_pos = flockSys.getPos();                                                
        uint * cellOcc = thrust::raw_pointer_cast(&d_cellOcc[0]);                                                                          
        thrust::fill(d_cellOcc.begin(), d_cellOcc.end(), 0);                                                    
        PointHashOperator hashOp(cellOcc);                                                                      
                                                                                                     
        thrust::transform(d_pos.begin(), d_pos.end(), d_hash.begin(), hashOp);                       
                                                                                                            
    }                                                                                                           