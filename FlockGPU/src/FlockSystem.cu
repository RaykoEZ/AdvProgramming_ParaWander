#include "hip/hip_runtime.h"
#include "FlockSystem.h"
#include "Hash.cuh"
#include "DebugUtil.cuh"
#include <random>
#include <iostream>
#include "FlockParams.cuh"
#include "FlockKernels.cuh"
#include "Random.cuh"


FlockSystem::FlockSystem(const uint &_numP, const float &_m, const float &_vMax, const float &_dt, const float &_rad, const float &_res)
{
    h_params = new FlockParams(_numP,_m,_vMax,_dt,_res);
    h_params->setNumBoids(_numP);
    if(_m > 0.0f)
    {
        h_params->setMass(_m);
        h_params->setInverseMass(1.0f/_m);

    } 
    else
    {
        h_params->setMass(DEFAULT_MASS);
        h_params->setInverseMass(DEFAULT_MASS_INV);
    }
    h_params->setVMax(_vMax);
    h_params->setTimeStep(_dt);
    h_spawnRad = _rad;
    h_frameCount = 0;
    h_init = false;
    

}

FlockSystem::~FlockSystem()
{
    clear();
    h_frameCount = 0;
    delete h_params;
}

void FlockSystem::init()
{
    clear();
    //h_params->init();
    /// resize vectors for future storage
    d_pos.resize(h_params->getNumBoids());
    d_v.resize(h_params->getNumBoids());
    d_vMax.resize(h_params->getNumBoids());
    d_target.resize(h_params->getNumBoids());
    d_col.resize(h_params->getNumBoids());

    d_angle.resize(h_params->getNumBoids());
    d_hash.resize(h_params->getNumBoids(),0);
    d_cellOcc.resize(h_params->getRes2(),0);
    d_scatterAddress.resize(h_params->getRes2());
    d_isThereCollision.resize(h_params->getNumBoids());

    thrust::fill(d_col.begin(), d_col.end(),make_float3(0.0f,255.0f,0.0f));
    thrust::fill(d_isThereCollision.begin(), d_isThereCollision.end(),false);

    
    prepareBoids(h_params->getNumBoids(), 0.1f,0.1f,
                                          0.9f,0.9f);




    //cudaErrorPrint();
    h_params->init();
}



void FlockSystem::tick()
{
    if(!h_init) return;




    /// We cast to raw ptr for kernel calls
    float3 * pos = thrust::raw_pointer_cast(&d_pos[0]);
    float3 * velocity = thrust::raw_pointer_cast(&d_v[0]);
    float3 * targetPos = thrust::raw_pointer_cast(&d_target[0]);
    float3 * colour = thrust::raw_pointer_cast(&d_col[0]);
    float * vMax = thrust::raw_pointer_cast(&d_vMax[0]);
    bool * collision = thrust::raw_pointer_cast(&d_isThereCollision[0]);
    float * angle = thrust::raw_pointer_cast(&d_angle[0]);
    uint * cellOcc = thrust::raw_pointer_cast(&d_cellOcc[0]);
    uint * scatter = thrust::raw_pointer_cast(&d_scatterAddress[0]);

    /// Set random floats for boid wandering search angle
    randomFloats(angle, h_params->getNumBoids(),h_frameCount);
    /// flush prior occupancy out and put new occupancy data in
    thrust::fill(d_cellOcc.begin(), d_cellOcc.end(), 0);
    PointHashOperator hashOp(cellOcc);
    thrust::transform(d_pos.begin(), d_pos.end(), d_hash.begin(), hashOp);




    //cudaErrorPrint();

    thrust::sort_by_key(
    d_hash.begin(),
    d_hash.end(),
    thrust::make_zip_iterator(thrust::make_tuple(d_pos.begin(),
                                                 d_v.begin(),
                                                 d_target.begin(),
                                                 d_angle.begin(),
                                                 d_vMax.begin()
    )));
                    



    //cudaErrorPrint();
    thrust::exclusive_scan(d_cellOcc.begin(), d_cellOcc.end(), d_scatterAddress.begin());
    //cudaErrorPrint();
    uint maxCellOcc = thrust::reduce(d_cellOcc.begin(), d_cellOcc.end(), 0, thrust::maximum<unsigned int>());

    /// define block dims to solve for ths frame
    uint blockSize = 32 * ceil(maxCellOcc / 32.0f);
    dim3 gridSize(h_params->getRes(), h_params->getRes());
    //std::cout<<"Res dump: "<< h_params->getRes()<<'\n';
    //d_threadIdxCheck.resize(blockSize);
    //d_blockIdxCheck.resize(h_params->getRes2());

    //uint * thread = thrust::raw_pointer_cast(&d_threadIdxCheck[0]);
    //uint * block = thrust::raw_pointer_cast(&d_blockIdxCheck[0]);
    /// We update boids in gpu below

    /// Spatial hash values, cell occupancy, memory scatter offset ( scatter addresses),
    /// positions and direction are already initialized, now we:
    /// - determine neighbourhood and collision flag
    /// - calculate target position and behaviour depending on collision flag
    /// - resolve forces
    /// - change colours if colliding

    /// Modifies:
    /// - collision flag
    /// - Boid Target Position (to average neighbourhood position)
    ///

    std::cout << "maxCellOcc=" << maxCellOcc << ", blockSize=" << blockSize << ", gridSize=" << h_params->getRes() << "^2\n";

    computeAvgNeighbourPos<<<gridSize, blockSize>>>(collision, targetPos, pos, cellOcc, scatter);
    hipDeviceSynchronize();

    //cudaErrorPrint();
    /// now we decide to wander if no collision, flee if there is collision
    genericBehaviour<<<gridSize,blockSize>>>(
                                               velocity,
                                               colour,
                                               targetPos,
                                               pos,
                                               collision,
                                               cellOcc,
                                               scatter,
                                               angle,
                                               vMax);
    hipDeviceSynchronize();

    //cudaErrorPrint();
    ++h_frameCount;
}

void FlockSystem::clear()
{
    d_pos.clear();
    d_v.clear();
    d_vMax.clear();
    d_target.clear();
    d_col.clear();
    d_angle.clear();
    d_isThereCollision.clear();
    d_hash.clear();
    d_cellOcc.clear();
    d_scatterAddress.clear();
    
    //d_threadIdxCheck.clear();
    //d_blockIdxCheck.clear();


}



void FlockSystem::prepareBoids(const float &_nBoids,
                               const float &_minX, const float &_minY,
                               const float &_maxX, const float &_maxY)
{

    float3 minCorner = make_float3(_minX, _minY, 0.0f);
    float3 maxCorner = make_float3(_maxX, _maxY, 0.0f);

    float3 diff = maxCorner - minCorner;
    float3 halfDiff = 0.5f * diff;
    float3 mid = make_float3(minCorner.x + halfDiff.x,minCorner.y + halfDiff.y,0.0f);


    std::random_device rd;
    std::mt19937_64 gen(rd());

    float3 quartDiff = 0.5f* halfDiff;
    float rad = length(quartDiff);
    float boidRad =0.5f * h_params->getInvRes();
    h_params->setCollisionRad(boidRad);
    std::uniform_real_distribution<float> spawnDis(0.1f, 0.9f);

    std::uniform_real_distribution<float> vDis(-1.0f, 1.0f);
    std::uniform_real_distribution<float> vMaxDis(1.0f, 10.0f);



    std::vector<float3> posHost;
    std::vector<float3> vHost;
    std::vector<float> vMaxHost;

    float3 pos;
    float3 v;
    for(unsigned int i = 0; i < _nBoids; ++i)
    {
        pos = make_float3(spawnDis(gen),spawnDis(gen), 0.0f);

        //std::cout<< pos.x<<", "<< pos.y<< ", "<< pos.z<<'\n';
        v = make_float3(vDis(gen), vDis(gen), 0.0f);
        posHost.push_back(pos);
        vHost.push_back(v);
        vMaxHost.push_back(vMaxDis(gen));
    }
    /// copy pos and velocity results to device vector
    thrust::copy(posHost.begin(),posHost.end(),d_pos.begin());
    thrust::copy(vHost.begin(),vHost.end(),d_v.begin());
    thrust::copy(vMaxHost.begin(),vMaxHost.end(),d_vMax.begin());

    h_init = true;


}
void FlockSystem::exportResult(std::vector<float3> &_posh, std::vector<float3> &_colh) const
{
    thrust::copy(d_col.begin(), d_col.end(), _colh.begin());
    thrust::copy(d_pos.begin(), d_pos.end(), _posh.begin());

    
    //std::vector<uint> hashH;
    //hashH.resize(h_params->getNumBoids());
    //thrust::copy(d_hash.begin(),d_hash.end(), hashH.begin());
    //std::vector<float3> vh;
    //vh.resize(h_params->getNumBoids());
    //std::vector<float3> targeth;
    //targeth.resize(h_params->getNumBoids());
    //thrust::copy(d_v.begin(), d_v.end(),vh.begin());
    //thrust::copy(d_target.begin(), d_target.end(),targeth.begin());

    //std::vector<bool> collisionh;
    //std::vector<float> angleh;
    //std::vector<uint> occh;
    //occh.resize(h_params->getRes2());
    //angleh.resize(h_params->getNumBoids());
    //collisionh.resize(h_params->getNumBoids());
    //thrust::copy(d_cellOcc.begin(), d_cellOcc.end(),occh.begin());
    //thrust::copy(d_isThereCollision.begin(), d_isThereCollision.end(),collisionh.begin());
    //thrust::copy(d_angle.begin(), d_angle.end(),angleh.begin());
    //std::cout<<"Num of Cells allocated: "<< h_params->getRes2() << '\n';
    //uint size = h_params->getNumBoids();
    
    //for(int i = 0; i< size;++i)
    //{
        //std::cout<< "Collision check: "<< collisionh[i] << '\n';
        //std::cout<< "Hash check: "<< hashH[i] << '\n';

        //std::cout<<"Angle Check: " << angleh[i] * 360.0f <<'\n';
        //std::cout<<"Pos Check: "<< _posh[i].x << ", " << _posh[i].y<< '\n';
        //std::cout<<"V Check: "<< vh[i].x << ", " << vh[i].y<< '\n';
        //std::cout<<"Target Check: "<< targeth[i].x << ", " << targeth[i].y<< '\n';

    //}
    
    //std::vector<uint> threadH;
    //std::vector<uint> blockH;
    //threadH.resize(d_threadIdxCheck.size());
    //blockH.resize(d_blockIdxCheck.size());

    //thrust::copy(d_threadIdxCheck.begin(),d_threadIdxCheck.end(),threadH.begin());
    //thrust::copy(d_blockIdxCheck.begin(),d_blockIdxCheck.end(),blockH.begin());

    //for(int i = 0; i< threadH.size();++i)
    //{
        
    //    std::cout<< "ThreadIdx check: "<< threadH[i]<<'\n';

        //std::cout<< "Occupancy check: "<< occh[i] << '\n'; 
    //}
    /*
    for(int i = 0; i< h_params->getRes2();++i)
    {
        
        //std::cout<< "BlockIdx check: "<< blockH[i].x << ", "<< blockH[i].y<<", "<<blockH[i].z<<'\n';

        std::cout<< "Occupancy check in blocks: "<< blockH[i] << '\n'; 
    }*/
    



}
