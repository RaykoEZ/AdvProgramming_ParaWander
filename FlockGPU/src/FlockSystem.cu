#include "hip/hip_runtime.h"
#include "FlockSystem.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include <thrust/device_vector.h>

#include <random>
#include <time.h>
#include <vector>
#include <iostream>

#include "FlockParams.cuh"

FlockSystem::FlockSystem(const unsigned int &_numP, const float &_m, const float &_vMax, const float &_dt)
{
    m_params = new FlockParams(_numP,_m,_vMax,_dt);

}

FlockSystem::~FlockSystem()
{
    clear();
    delete m_params;
}

void FlockSystem::init(const unsigned int &_numP, const unsigned int &_res)
{
    clear();

}


void FlockSystem::setup(const unsigned int &_numP, const unsigned int &_res)
{



}

void FlockSystem::tick(const float &_dt)
{

}

void FlockSystem::clear()
{
    m_pos.clear();
    m_v.clear();
    m_target.clear();
    m_isThereCollision.clear();
    m_hash.clear();
    m_cellOcc.clear();
    m_scatterAddress.clear();


}
void FlockSystem::createSpawnCircle(const float &_rad, const float3 &_origin)
{

}
