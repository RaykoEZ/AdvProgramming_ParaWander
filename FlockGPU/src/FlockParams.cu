#include "FlockParams.cuh"
#include <iostream>

__constant__ FlockData paramData;

FlockParams::FlockParams(const unsigned int &_numB,
                              const float &_m,
                              const float &_vMax,
                              const float &_dt,
                              const unsigned int &_res)
{
    setNumBoids(_numB);
    setMass(_m);
    setInverseMass(_m);
    setVMax(_vMax);
    setTimeStep(_dt);
    setRes(_res);
    
}

void FlockParams::init()
{
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(paramData), &m_data, sizeof(FlockData));
    if (err != hipSuccess)
    {
        std::cerr << "Copy to symbol params (size=" << sizeof(FlockParams) << ") failed! Reason: " << hipGetErrorString(err) << "\n";
        exit(0);
    }

}

void FlockParams::setNumBoids(const unsigned int &_numB)
{
    m_data.m_numBoids=_numB;
}

void FlockParams::setMass(const float &_m)
{

    m_data.m_mass=_m;
}

void FlockParams::setInverseMass(const float &_m)
{
    if(_m > 0)
    {
        m_data.m_invMass = 1.0f/_m;
    }
    else return;

}

void FlockParams::setVMax(const float &_vMax)
{
    m_data.m_vMax=_vMax;
}

void FlockParams::setTimeStep(const float &_dt)
{

    m_data.m_dt=_dt;
}

void FlockParams::setRes(const unsigned int &_res)
{
    m_data.m_res = _res;
    m_data.m_res2 = _res * _res;
    m_data.m_invRes = 1.0f/_res;
    m_data.m_invRes2 = m_data.m_invRes * m_data.m_invRes;

}

void FlockParams::setCollisionRad(const float &_rad)
{ 
    m_data.m_collisionRad = _rad;
}