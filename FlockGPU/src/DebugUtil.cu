#include "DebugUtil.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>


void cudaErrorPrint()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "Thrust allocation failed, error " << hipGetErrorString(err) << "\n";
        
    }

}