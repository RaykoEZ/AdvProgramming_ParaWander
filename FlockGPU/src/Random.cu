#include "Random.cuh"
#include <hip/hip_runtime.h>
#include <hiprand.h>

void randomFloats(float * &_out, const size_t _n, const uint &_seed)
{
    hiprandGenerator_t rng;

    hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT);

    int seed = time(NULL)+ _seed;
    hiprandSetPseudoRandomGeneratorSeed(rng, seed);

    hiprandGenerateUniform(rng, _out ,_n);

    hiprandDestroyGenerator(rng);

}

