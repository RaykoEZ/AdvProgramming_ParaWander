#include "Random.cuh"
#include <hip/hip_runtime.h>
#include <hiprand.h>

void randomFloats(float *&_out, const size_t _n)
{
    hiprandGenerator_t rng;

    hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT);

    hiprandSetPseudoRandomGeneratorSeed(rng, time(NULL));

    hiprandGenerateUniform(rng, _out ,_n);

    hiprandDestroyGenerator(rng);

}

